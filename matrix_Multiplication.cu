#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#define BLOCK_SIZE 16

/*
 * prints matrices
 * Because matrices filled with dummy 0s function takes 3 dim arguments:
 *      actual x and y dimension and dim as big square matrix's dimension
 */
void print_matrices(float* matrix, char* file_Name, int x_dim, int y_dim, int dim)
{
    std::ofstream outFile;
    outFile.open (file_Name);

    outFile << std::fixed;
    outFile << std::setprecision(2);

    for (int i = 0; i < x_dim; i++) {

        for (int j = 0; j < y_dim; j++) {
            outFile << matrix[i * dim + j] << " ";
        }
        outFile << std::endl;
    }
}

//naive CPU matrix multiplication code
//because of its simplicity directly taken from web
//it multiplies square matrices
__host__ void cpu_matrix_mult(float *h_a, float *h_b, float *h_result, int m) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < m; ++j)
        {
            float tmp = 0.0;
            for (int h = 0; h < m; ++h)
            {
                tmp += h_a[i * m + h] * h_b[h * m + j];
            }
            h_result[i * m + j] = tmp;
        }
    }
}

//this function is for filling the matrices with cos and sin values randomly
//I transform the matrices to square matrix in order to perform better multiplication
__host__ int fill(float **Lmatrix, float **Rmatrix, int LdimX, int LdimY, int RdimX, int RdimY) {

    int sqr_dim_X, sqr_dim_Y, size;

    sqr_dim_X = RdimX;
    if (LdimX > RdimX) {
        sqr_dim_X = LdimX;
    }

    sqr_dim_Y = RdimY;
    if (LdimY > RdimY) {
        sqr_dim_Y = LdimY;
    }

    size = sqr_dim_Y;
    if (sqr_dim_X > sqr_dim_Y) {
        size = sqr_dim_X;
    }

    int temp = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1);
    size = temp * BLOCK_SIZE;

    size_t pt_size = size * size * sizeof(float);

    *Lmatrix = (float *) malloc(pt_size);
    *Rmatrix = (float *) malloc(pt_size);

    memset(*Lmatrix, 0, pt_size);
    memset(*Rmatrix, 0, pt_size);

    for (int i = 0; i < LdimX; i++) {
        for (int j = 0; j < LdimY; j++) {
            int dummy = size * i + j;
            (*Lmatrix)[dummy] = sinf(dummy);
        }
    }
    for (int i = 0; i < RdimX; i++) {
        for (int j = 0; j < RdimY; j++) {
            int dummy = size * i + j;
            (*Rmatrix)[dummy] = cosf(dummy);
        }
    }
    return size;
}

// Kernel that executes on the CUDA device
/* left: left operand
 * right: right operand
 * res : result array
 * dim: M dimension of MxM matrix
 * Blok_size: defines block size
 *
 * this function divides the matrices to tiles and load those tiles to shared memory
 * After loading to shared memory it function multiplies with the corresponding tile of other matrix
 * After finishing multiplication of 1 row and 1 column by collecting results of different tiles
 * it stores the result in global memory
 * Function has coalesced access to the global memory and prevent bank conflict
 */
__global__ void multiply(float *left, float *right, float *res, int dim) {

    int i,j;
    float temp = 0;

    __shared__ float Left_shared_t [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

    // Row i of matrix left
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

        // Column j of matrix left
        j = tileNUM * BLOCK_SIZE + threadIdx.x;
        i = tileNUM * BLOCK_SIZE + threadIdx.y;
        // Load left[i][j] to shared mem

        Left_shared_t[threadIdx.y][threadIdx.x] = left[row * dim + j];// Coalesced access
        // Load right[i][j] to shared mem

        Right_shared_t[threadIdx.y][threadIdx.x] = right[i * dim + col]; // Coalesced access
        // Synchronize before computation
        __syncthreads();

        // Accumulate one tile of res from tiles of left and right in shared mem
        for (int k = 0; k < BLOCK_SIZE; k++) {

            temp += Left_shared_t[threadIdx.y][k] * Right_shared_t[k][threadIdx.x]; //no shared memory bank conflict
        }
        // Synchronize
        __syncthreads();
    }
    // Store accumulated value to res
    res[row * dim + col] = temp;
}

// main routine that executes on the host
int main(void)
{
    //size of the vectors to be processed  and matrix dimensions
    int Left_matrix_x, Left_matrix_y, Right_matrix_x, Right_matrix_y, Left_vector_size, Right_vector_size;

    float *Left_Vector_h, *Right_Vector_h, *Left_Vector_d, *Right_Vector_d, *Res_h, *Res_d, *CPU;  // Pointer to host & device arrays

    printf("Enter m n n k :\n");

    scanf("%d %d %d %d",&Left_matrix_x,&Left_matrix_y,&Right_matrix_x,&Right_matrix_y); // input matrix dimensions are taken

    int dim = fill(&Left_Vector_h, &Right_Vector_h, Left_matrix_x, Left_matrix_y, Right_matrix_x, Right_matrix_y); //fills the matrices with random values

    print_matrices(Left_Vector_h,"Input_LHS",Left_matrix_x,Left_matrix_y,dim);
    print_matrices(Right_Vector_h,"Input_RHS",Right_matrix_x,Right_matrix_y,dim);

    size_t vector_size;
    vector_size = dim*dim * sizeof(float);

    Res_h = (float *) malloc(vector_size); // Allocate array on host for result
    CPU = (float *) malloc(vector_size);// Allocate array on host for CPU_matrix_multiplication result

    hipMalloc((void **) &Left_Vector_d, vector_size);     // Allocate array on device for LHS operand
    hipMalloc((void **) &Right_Vector_d, vector_size);   // Allocate array on device for RHS operand but this is vector 1xN
    hipMalloc((void **) &Res_d, vector_size);     // Allocate array on device for result

    hipMemcpy(Left_Vector_d, Left_Vector_h, vector_size, hipMemcpyHostToDevice);      // copy values to device
    hipMemcpy(Right_Vector_d, Right_Vector_h, vector_size, hipMemcpyHostToDevice);   // copy values to device

    //Block dimension is directly from block_size
    dim3 Block_dim(BLOCK_SIZE, BLOCK_SIZE);
    //Grid dimension is found by dividing matrix dimension to block_size
    dim3 Grid_dim(dim / BLOCK_SIZE, dim / BLOCK_SIZE);

    //commented out the functions which helps to calculate time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //kernel call
    multiply << < Grid_dim, Block_dim >> > (Left_Vector_d, Right_Vector_d, Res_d, dim);

    //commented out the functions which helps to calculate time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Retrieve result from device and store it in host array
    hipMemcpy(Res_h, Res_d, vector_size, hipMemcpyDeviceToHost);

    clock_t begin = clock();

    cpu_matrix_mult(Left_Vector_h,Right_Vector_h,CPU,dim); //matrix multiplication on cpu

    clock_t end = clock();
    double time_spent = (double)1000*(end - begin) / CLOCKS_PER_SEC;

    //commented out the functions which helps to calculate time
    printf("GPU time= %f ms\n", et);

    printf("CPU time= %lf ms\n", time_spent);

    //Prints the results
    print_matrices(Res_h,"GPU_out",Left_matrix_x,Right_matrix_y,dim);
    print_matrices(CPU,"CPU_out",Left_matrix_x,Right_matrix_y,dim);

    bool equal = true;
    for (int i=0;i< Left_matrix_x && equal;i++){
        for (int j = 0; j < Right_matrix_y && equal; j++) {
            if (abs(Res_h[i*dim+j]-CPU[i*dim+j]) > 0.001)
            {
                equal = false;
                printf("NOT EQUAL\n");
            }
        }
    }
    if (equal)
    {
        std::cout<<"Results are equal!"<<std::endl;
    }
    else
    {
        std::cout<<"Results are NOT equal!"<<std::endl;
    }

    // Cleanup
    free(Left_Vector_h);
    free(Right_Vector_h);
    free(Res_h);
    free(CPU);
    hipFree(Left_Vector_d);
    hipFree(Right_Vector_d);
    hipFree(Res_d);
}
